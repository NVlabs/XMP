#include "hip/hip_runtime.h"
/***
Copyright (c) 2015, NVIDIA CORPORATION.  All rights reserved.

Permission is hereby granted, free of charge, to any person obtaining a
copy of this software and associated documentation files (the "Software"),
to deal in the Software without restriction, including without limitation
the rights to use, copy, modify, merge, publish, distribute, sublicense,
and/or sell copies of the Software, and to permit persons to whom the
Software is furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in
all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS
IN THE SOFTWARE.
***/

#include <operators.h>

//prevent instantiation of these here....
extern template xmpError_t internalPowmRegMP<128,4,4,0,0>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmRegMP<128,4,8,0,0>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmRegMP<128,4,12,0,0>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmRegMP<128,4,16,0,0>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,4,1>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,2,2>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,8,1>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,4,2>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,2,4>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,4,3>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,2,6>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,16,1>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,8,2>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,4,4>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,2,8>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,8,3>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,4,6>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,32,1>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,16,2>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,8,4>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,4,8>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,16,3>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,8,6>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,32,2>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,16,4>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,8,8>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,32,3>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,16,6>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,32,4>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,16,8>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,32,6>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmWarpDistributedMP<128,4,32,8>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t* );
extern template xmpError_t internalPowmDigitMP<128,4,8>(xmpHandle_t,xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, const xmpIntegers_t, uint32_t, uint32_t, uint32_t*, uint32_t*);

//uint32_t xmpPowmPrecisions[]={128,256,512,768,1024,1536,2048,3072,4096,6144,8192};
uint32_t xmpPowmPrecisions[]={128,256,512,768,1024,1536,2048,3072,4096};
uint32_t xmpPowmPrecisionsCount = sizeof(xmpPowmPrecisions)/sizeof(uint32_t);

xmpPowmAlgorithm xmpPowmAlgorithms[] = {
  //ThreeN
  xmpPowmAlgorithm(xmpAlgorithmRegMP,(xmpPowmFunc)internalPowmRegMP<128,4,4,0,0>,1,128),
  xmpPowmAlgorithm(xmpAlgorithmRegMP,(xmpPowmFunc)internalPowmRegMP<128,4,8,0,0>,129,256),
  xmpPowmAlgorithm(xmpAlgorithmRegMP,(xmpPowmFunc)internalPowmRegMP<128,4,12,0,0>,257,384),
  xmpPowmAlgorithm(xmpAlgorithmRegMP,(xmpPowmFunc)internalPowmRegMP<128,4,16,0,0>,385,512),

  //Distributed
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,4,1>,1,128),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,2,2>,1,128),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,8,1>,129,256),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,4,2>,129,256),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,2,4>,129,256),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,4,3>,257,384),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,2,6>,257,384),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,16,1>,385,512),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,8,2>,385,512),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,4,4>,385,512),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,2,8>,385,512),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,8,3>,513,768),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,4,6>,513,768),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,32,1>,767,1024),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,16,2>,767,1024),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,8,4>,767,1024),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,4,8>,767,1024),
#if 1
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,16,3>,1025,1536),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,8,6>,1025,1536),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,32,2>,1537,2048),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,16,4>,1537,2048),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,8,8>,1537,2048),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,32,3>,2049,3072),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,16,6>,2049,3072),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,32,4>,3073,4096),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,16,8>,3073,4096),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,32,6>,4097,6144),
  xmpPowmAlgorithm(xmpAlgorithmDistributedMP,(xmpPowmFunc)internalPowmWarpDistributedMP<128,4,32,8>,6145,8192),
#endif
  //Digitized
  xmpPowmAlgorithm(xmpAlgorithmDigitMP,(xmpPowmFunc)internalPowmDigitMP<128,4,8>,512,uint32_t(-1)),
};
uint32_t xmpPowmAlgorithmsCount = sizeof(xmpPowmAlgorithms)/sizeof(xmpPowmAlgorithm);

struct LaunchParameters
{
  int32_t alg_index;
  uint32_t count;
  LaunchParameters(int32_t alg_index, uint32_t count) : alg_index(alg_index), count(count) {}
  LaunchParameters() : alg_index(-1), count(0) {}
};

#include <tune/tune_maxwell.h>
#include <tune/tune_kepler.h>


LaunchParameters getPowmLaunchParameters(xmpHandle_t handle, uint32_t precision, uint32_t count, xmpAlgorithm_t alg ) {
  const Latency * lookup;
  uint32_t tbl_count;
  
  LaunchParameters params;

  float max_throughput=0;
  params.alg_index=-1;

  int idx;
  if(precision>xmpPowmPrecisions[xmpPowmPrecisionsCount-1]) {
    if(alg!=xmpAlgorithmDefault && alg!=xmpAlgorithmDigitMP)
      return LaunchParameters();

    LaunchParameters params;
    //force it to digitized
    params.alg_index=xmpPowmAlgorithmsCount-1;
    //run at full count (assumes count isn't so big that it overflows CUDA)
    params.count=count;
    return params;
  }

  idx=0;
  for(int i=0;i<xmpPowmPrecisionsCount;i++) {
    if(precision<=xmpPowmPrecisions[i]) {
      idx=i;
      break;
    }
  }

  if(handle->arch<50) {
    lookup=powm_tbl_kepler[idx];
    count=powm_tbl_kepler_counts[idx];
  }else {
    lookup=powm_tbl_maxwell[idx];
    tbl_count=powm_tbl_maxwell_counts[idx];
  }

  for(int i=0;i<tbl_count;i++) {
    Latency lat=lookup[i];

    if(alg!=xmpAlgorithmDefault && alg!=xmpPowmAlgorithms[lat.alg_index].alg) {
      continue;
    }

    uint32_t lcount = MIN(lat.instances_per_sm*handle->smCount,count);
    float throughput = lcount/lat.latency;
    if(throughput>max_throughput) {
      params.alg_index=lat.alg_index;
      params.count=lcount;
      max_throughput=throughput;
    }
  }
  //hueristic:  launch as many waves of this size as possible,  we see better proformance with mulitple waves than single waves.  
  //This is likely due to launch latency.
  uint32_t waves = count/params.count;  //number of waves that could fit in the full size
  params.count = waves*params.count;    //scale count by that many waves
  //printf("POWM: precison: %d, count: %d, lcount: %d,  alg: %d\n", precision, count, params.count, params.alg_index);
  return params;
}

template<class T>
inline void configureActiveBlocks(xmpHandle_t handle, dim3 &blocks, dim3 threads, T *kernel) {
  int         maxBlocks;
  hipError_t error;

  if(GSL) {
    error=hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxBlocks, kernel, threads.x, 0);
    if(error==hipSuccess && blocks.x>maxBlocks*handle->smCount)
      blocks.x=maxBlocks*handle->smCount;
  }
}


//computes s=a+b
xmpError_t XMPAPI xmpIntegersAdd(xmpHandle_t handle, xmpIntegers_t s, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  xmpError_t error=xmpIntegersAddAsync(handle,s,a,b,count);
  if(error!=xmpErrorSuccess)
    return error;
  XMP_SET_DEVICE(handle);
  hipStreamSynchronize(handle->stream);
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
xmpError_t XMPAPI xmpIntegersAddAsync(xmpHandle_t handle, xmpIntegers_t s, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  dim3            blocks(DIV_ROUND_UP(count, GEOMETRY)), threads(GEOMETRY);
  int             device=handle->device;
  add_arguments_t add_arguments;
  xmpExecutionPolicy_t policy=handle->policy;

  if(s->device!=device || a->device!=device || b->device!=device)
    return xmpErrorInvalidDevice;

  if(count==0)
    return xmpErrorSuccess;

  XMP_SET_DEVICE(handle);

  if(s->count<count)
    return xmpErrorInvalidCount;

  if(policy->indices[0] && policy->indices_count[0]<count)
    return xmpErrorInvalidCount;

  if(s->precision<MAX(a->precision,b->precision))
    return xmpErrorInvalidPrecision;

  XMP_CHECK_CUDA();

  a->requireFormat(handle, xmpFormatStrided);
  b->requireFormat(handle, xmpFormatStrided);

  xmpAlgorithm_t alg = policy->algorithm;
  if(alg==xmpAlgorithmDefault) {
    if(a->precision<=512 && b->precision<=512)
      alg = xmpAlgorithmRegMP;
    else
      alg = xmpAlgorithmDigitMP;
  }

  // package up the arguments
  add_arguments.out_data=s->slimbs;
  add_arguments.out_len=DIV_ROUND_UP(s->precision, 32);
  add_arguments.out_stride=s->stride;
  add_arguments.a_data=a->slimbs;
  add_arguments.a_len=DIV_ROUND_UP(a->precision, 32);
  add_arguments.a_stride=a->stride;
  add_arguments.a_count=a->count;
  add_arguments.b_data=b->slimbs;
  add_arguments.b_len=DIV_ROUND_UP(b->precision, 32);
  add_arguments.b_stride=b->stride;
  add_arguments.b_count=b->count;
  add_arguments.out_indices=policy->indices[0];
  add_arguments.a_indices=policy->indices[1];
  add_arguments.b_indices=policy->indices[2];
  add_arguments.a_indices_count=policy->indices_count[1];
  add_arguments.b_indices_count=policy->indices_count[2];

  //if s is in-place we need to work in scratch memory
  bool inplace = (s==a || s==b);
  size_t out_size=s->stride*s->nlimbs*sizeof(xmpLimb_t);

  if(inplace) {
    xmpError_t error;
    error=xmpSetNecessaryScratchSize(handle, out_size);
    if(error!=xmpErrorSuccess)
      return error;
    add_arguments.out_data=(xmpLimb_t*)handle->scratch;
  }

  if(alg==xmpAlgorithmRegMP)  {
    if(a->precision<=128 && b->precision<=128) {
      configureActiveBlocks(handle, blocks, threads, regmp_add_kernel<GSL, 4>);
      regmp_add_kernel<GSL, 4><<<blocks, threads, 0, handle->stream>>>(add_arguments, count);
      goto done;
    }
    else if(a->precision<=256 && b->precision<=256) {
      configureActiveBlocks(handle, blocks, threads, regmp_add_kernel<GSL, 8>);
      regmp_add_kernel<GSL, 8><<<blocks, threads, 0, handle->stream>>>(add_arguments, count);
      goto done;
    }
    else if(a->precision<=384 && b->precision<=384) {
      configureActiveBlocks(handle, blocks, threads, regmp_add_kernel<GSL, 12>);
      regmp_add_kernel<GSL, 12><<<blocks, threads, 0, handle->stream>>>(add_arguments, count);
      goto done;
    }
    else if(a->precision<=512 && b->precision<=512) {
      configureActiveBlocks(handle, blocks, threads, regmp_add_kernel<GSL, 16>);
      regmp_add_kernel<GSL, 16><<<blocks, threads, 0, handle->stream>>>(add_arguments, count);
      goto done;
    }
  }
  else if(alg==xmpAlgorithmDigitMP) {
    configureActiveBlocks(handle, blocks, threads, digitmp_add_kernel<GSL, DIGIT>);
    digitmp_add_kernel<GSL, DIGIT><<<blocks, threads, 0, handle->stream>>>(add_arguments, count);
    goto done;
  }

  //this is only reached if they requested an unsupported algorithm
  return xmpErrorUnsupported;

done:

  if(inplace) {
    hipMemcpyAsync(s->slimbs,add_arguments.out_data,out_size,hipMemcpyDeviceToDevice,handle->stream);
  }

  s->setFormat(xmpFormatStrided);

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

//computes d=a-b
xmpError_t XMPAPI xmpIntegersSub(xmpHandle_t handle, xmpIntegers_t d, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  xmpError_t error=xmpIntegersSubAsync(handle,d,a,b,count);
  if(error!=xmpErrorSuccess)
    return error;
  XMP_SET_DEVICE(handle);
  hipStreamSynchronize(handle->stream);
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
xmpError_t XMPAPI xmpIntegersSubAsync(xmpHandle_t handle, xmpIntegers_t d, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  dim3            blocks(DIV_ROUND_UP(count, GEOMETRY)), threads(GEOMETRY);
  int             device=handle->device;
  sub_arguments_t sub_arguments;
  xmpExecutionPolicy_t policy=handle->policy;

  if(d->device!=device || a->device!=device || b->device!=device)
    return xmpErrorInvalidDevice;

  if(count==0)
    return xmpErrorSuccess;

  XMP_SET_DEVICE(handle);

  if(d->count<count)
    return xmpErrorInvalidCount;

  if(policy->indices[0] && policy->indices_count[0]<count)
    return xmpErrorInvalidCount;

  if(d->precision<MAX(a->precision,b->precision))
    return xmpErrorInvalidPrecision;

  a->requireFormat(handle, xmpFormatStrided);
  b->requireFormat(handle, xmpFormatStrided);

  xmpAlgorithm_t alg = policy->algorithm;
  if(alg==xmpAlgorithmDefault) {
    if(a->precision<=512 && b->precision<512)
      alg = xmpAlgorithmRegMP;
    else
      alg = xmpAlgorithmDigitMP;
  }

  // package up the arguments
  sub_arguments.out_data=d->slimbs;
  sub_arguments.out_len=DIV_ROUND_UP(d->precision, 32);
  sub_arguments.out_stride=d->stride;
  sub_arguments.a_data=a->slimbs;
  sub_arguments.a_len=DIV_ROUND_UP(a->precision, 32);
  sub_arguments.a_stride=a->stride;
  sub_arguments.a_count=a->count;
  sub_arguments.b_data=b->slimbs;
  sub_arguments.b_len=DIV_ROUND_UP(b->precision, 32);
  sub_arguments.b_stride=b->stride;
  sub_arguments.b_count=b->count;
  sub_arguments.out_indices=policy->indices[0];
  sub_arguments.a_indices=policy->indices[1];
  sub_arguments.b_indices=policy->indices[2];
  sub_arguments.a_indices_count=policy->indices_count[1];
  sub_arguments.b_indices_count=policy->indices_count[2];

  //if d is indexed and in-place we need to work in scratch memory
  bool inplace=(d==a || d==b);
  size_t out_size=d->stride*d->nlimbs*sizeof(xmpLimb_t);

  if(inplace) {
    xmpError_t error;
    error=xmpSetNecessaryScratchSize(handle, out_size);
    if(error!=xmpErrorSuccess)
      return error;
    sub_arguments.out_data=(xmpLimb_t*)handle->scratch;
  }

  if(alg==xmpAlgorithmRegMP) {
    if(a->precision<=128 && b->precision<=128) {
      configureActiveBlocks(handle, blocks, threads, regmp_sub_kernel<GSL, 4>);
      regmp_sub_kernel<GSL, 4><<<blocks, threads, 0, handle->stream>>>(sub_arguments, count);
      goto done;
    }
    else if(a->precision<=256 && b->precision<=256) {
      configureActiveBlocks(handle, blocks, threads, regmp_sub_kernel<GSL, 8>);
      regmp_sub_kernel<GSL, 8><<<blocks, threads, 0, handle->stream>>>(sub_arguments, count);
      goto done;
    }
    else if(a->precision<=384 && b->precision<=384) {
      configureActiveBlocks(handle, blocks, threads, regmp_sub_kernel<GSL, 12>);
      regmp_sub_kernel<GSL, 12><<<blocks, threads, 0, handle->stream>>>(sub_arguments, count);
      goto done;
    }
    else if(a->precision<=512 && b->precision<=512) {
      configureActiveBlocks(handle, blocks, threads, regmp_sub_kernel<GSL, 16>);
      regmp_sub_kernel<GSL, 16><<<blocks, threads, 0, handle->stream>>>(sub_arguments, count);
      goto done;
    }
  }
  else if(alg==xmpAlgorithmDigitMP){
    configureActiveBlocks(handle, blocks, threads, digitmp_sub_kernel<GSL, DIGIT>);
    digitmp_sub_kernel<GSL, DIGIT><<<blocks, threads, 0, handle->stream>>>(sub_arguments, count);
    goto done;
  }
  //this is only reached if they requested an unsupported algorithm
  return xmpErrorUnsupported;
done:
  if(inplace) {
    hipMemcpyAsync(d->slimbs,sub_arguments.out_data,out_size,hipMemcpyDeviceToDevice,handle->stream);
  }

  d->setFormat(xmpFormatStrided);

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
//computes p=a*a -- private but called from xmpIntegersMulAsync
xmpError_t XMPAPI xmpIntegersSqrAsync(xmpHandle_t handle, xmpIntegers_t p, const xmpIntegers_t a, uint32_t count) {
  dim3            blocks(DIV_ROUND_UP(count, GEOMETRY)), threads(GEOMETRY);
  int             device=handle->device;
  sqr_arguments_t sqr_arguments;
  xmpExecutionPolicy_t policy=handle->policy;

  if(p->device!=device || a->device!=device)
    return xmpErrorInvalidDevice;

  if(count==0)
    return xmpErrorSuccess;

  XMP_SET_DEVICE(handle);

  if(p->count<count)
    return xmpErrorInvalidCount;

  if(policy->indices[0] && policy->indices_count[0]<count)
    return xmpErrorInvalidCount;

  if(p->precision<a->precision)
    return xmpErrorInvalidPrecision;

  bool       inplace=(p==a);
  size_t     out_size=p->stride*p->nlimbs*sizeof(xmpLimb_t);
  xmpLimb_t *dst=p->slimbs;

  if(inplace) {
    xmpError_t error;

    error=xmpSetNecessaryScratchSize(handle, out_size);
    if(error!=xmpErrorSuccess)
      return error;
    dst=(xmpLimb_t*)handle->scratch;
  }

  a->requireFormat(handle, xmpFormatStrided);

  xmpAlgorithm_t alg = policy->algorithm;
  if(alg==xmpAlgorithmDefault) {
    if(a->precision<=512)
      alg = xmpAlgorithmRegMP;
    else
      alg = xmpAlgorithmDigitMP;
  }

  // package up the arguments
  sqr_arguments.out_data=dst;
  sqr_arguments.out_len=DIV_ROUND_UP(p->precision, 32);
  sqr_arguments.out_stride=p->stride;
  sqr_arguments.a_data=a->slimbs;
  sqr_arguments.a_len=DIV_ROUND_UP(a->precision, 32);
  sqr_arguments.a_stride=a->stride;
  sqr_arguments.a_count=a->count;
  sqr_arguments.out_indices=policy->indices[0];
  sqr_arguments.a_indices=policy->indices[1];
  sqr_arguments.a_indices_count=policy->indices_count[1];

  if(alg==xmpAlgorithmRegMP) {
    if(a->precision<=64) {
      configureActiveBlocks(handle, blocks, threads, regmp_sqr_kernel<GSL, 2>);
      regmp_sqr_kernel<GSL, 2><<<blocks, threads, 0, handle->stream>>>(sqr_arguments, count);
      goto done;
    }
    else if(a->precision<=128) {
      configureActiveBlocks(handle, blocks, threads, regmp_sqr_kernel<GSL, 4>);
      regmp_sqr_kernel<GSL, 4><<<blocks, threads, 0, handle->stream>>>(sqr_arguments, count);
      goto done;
    }
    else if(a->precision<=256) {
      configureActiveBlocks(handle, blocks, threads, regmp_sqr_kernel<GSL, 8>);
      regmp_sqr_kernel<GSL, 8><<<blocks, threads, 0, handle->stream>>>(sqr_arguments, count);
      goto done;
    }
    else if(a->precision<=384) {
      configureActiveBlocks(handle, blocks, threads, regmp_sqr_kernel<GSL, 12>);
      regmp_sqr_kernel<GSL, 12><<<blocks, threads, 0, handle->stream>>>(sqr_arguments, count);
      goto done;
    }
    else if(a->precision<=512) {
      configureActiveBlocks(handle, blocks, threads, regmp_sqr_kernel<GSL, 16>);
      regmp_sqr_kernel<GSL, 16><<<blocks, threads, 0, handle->stream>>>(sqr_arguments, count);
      goto done;
    }
  }
  else if(alg==xmpAlgorithmDigitMP) {
    configureActiveBlocks(handle, blocks, threads, digitmp_sqr_kernel<GSL, DIGIT>);
    digitmp_sqr_kernel<GSL, DIGIT><<<blocks, threads, 0, handle->stream>>>(sqr_arguments, count);
    goto done;
  }

  //this is only reached if they requested an unsupported algorithm
  return xmpErrorUnsupported;
done:

  if(inplace)
    hipMemcpyAsync(p->slimbs,dst,out_size,hipMemcpyDeviceToDevice,handle->stream);

  p->setFormat(xmpFormatStrided);

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

//computes p=a*b
xmpError_t XMPAPI xmpIntegersMul(xmpHandle_t handle, xmpIntegers_t p, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  xmpError_t error=xmpIntegersMulAsync(handle,p,a,b,count);
  if(error!=xmpErrorSuccess)
    return error;
  XMP_SET_DEVICE(handle);
  hipStreamSynchronize(handle->stream);
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
xmpError_t XMPAPI xmpIntegersMulAsync(xmpHandle_t handle, xmpIntegers_t p, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  dim3            blocks(DIV_ROUND_UP(count, GEOMETRY)), threads(GEOMETRY);
  int             device=handle->device;
  xmpIntegers_t   l, s;
  mul_arguments_t mul_arguments;
  xmpExecutionPolicy_t policy=handle->policy;

  if(a==b && policy->indices[1]==NULL && policy->indices[2]==NULL) {
    return xmpIntegersSqrAsync(handle, p, a, count);
  }

  if(p->device!=device || a->device!=device || b->device!=device)
    return xmpErrorInvalidDevice;

  if(count==0)
    return xmpErrorSuccess;

  XMP_SET_DEVICE(handle);

  if(p->count<count)
    return xmpErrorInvalidCount;

  if(policy->indices[0] && policy->indices_count[0]<count)
    return xmpErrorInvalidCount;

  if(p->precision<MAX(a->precision,b->precision))
    return xmpErrorInvalidPrecision;

  if(a->precision>=b->precision) {
    l=a;
    s=b;
  }
  else {
    l=b;
    s=a;
  }

  bool       inplace=(p==a || p==b);
  size_t     out_size=p->stride*p->nlimbs*sizeof(xmpLimb_t);
  xmpLimb_t *dst=p->slimbs;

  if(inplace) {
    xmpError_t error=xmpSetNecessaryScratchSize(handle, out_size);
    if(error!=xmpErrorSuccess)
      return error;
    dst=(xmpLimb_t*)handle->scratch;
  }

  l->requireFormat(handle, xmpFormatStrided);
  s->requireFormat(handle, xmpFormatStrided);

  xmpAlgorithm_t alg = policy->algorithm;
  if(alg==xmpAlgorithmDefault) {
    if(l->precision<=512 && s->precision<512)
      alg = xmpAlgorithmRegMP;
    else
      alg = xmpAlgorithmDigitMP;
  }


  // package up the arguments
  mul_arguments.out_data=dst;
  mul_arguments.out_len=DIV_ROUND_UP(p->precision, 32);
  mul_arguments.out_stride=p->stride;
  mul_arguments.a_data=l->slimbs;
  mul_arguments.a_len=DIV_ROUND_UP(l->precision, 32);
  mul_arguments.a_stride=l->stride;
  mul_arguments.a_count=l->count;
  mul_arguments.b_data=s->slimbs;
  mul_arguments.b_len=DIV_ROUND_UP(s->precision, 32);
  mul_arguments.b_stride=s->stride;
  mul_arguments.b_count=s->count;
  mul_arguments.out_indices=policy->indices[0];
  mul_arguments.a_indices=policy->indices[1];
  mul_arguments.b_indices=policy->indices[2];
  mul_arguments.a_indices_count=policy->indices_count[1];
  mul_arguments.b_indices_count=policy->indices_count[2];

  if(alg==xmpAlgorithmRegMP) {
    // multiply is a very common operator, so we have many sizes
    if(l->precision<=64 && s->precision<=64) {
      configureActiveBlocks(handle, blocks, threads, regmp_mul_kernel<GSL, 2, 2>);
      regmp_mul_kernel<GSL, 2, 2><<<blocks, threads, 0, handle->stream>>>(mul_arguments, count);
      goto done;
    }
    else if(l->precision<=128 && s->precision<=64) {
      configureActiveBlocks(handle, blocks, threads, regmp_mul_kernel<GSL, 4, 2>);
      regmp_mul_kernel<GSL, 4, 2><<<blocks, threads, 0, handle->stream>>>(mul_arguments, count);
      goto done;
    }
    else if(l->precision<=192 && s->precision<=64) {
      configureActiveBlocks(handle, blocks, threads, regmp_mul_kernel<GSL, 6, 2>);
      regmp_mul_kernel<GSL, 6, 2><<<blocks, threads, 0, handle->stream>>>(mul_arguments, count);
      goto done;
    }
    else if(l->precision<=256 && s->precision<=64) {
      configureActiveBlocks(handle, blocks, threads, regmp_mul_kernel<GSL, 8, 2>);
      regmp_mul_kernel<GSL, 8, 2><<<blocks, threads, 0, handle->stream>>>(mul_arguments, count);
      goto done;
    }
    else if(l->precision<=384 && s->precision<=64) {
      configureActiveBlocks(handle, blocks, threads, regmp_mul_kernel<GSL, 12, 2>);
      regmp_mul_kernel<GSL, 12, 2><<<blocks, threads, 0, handle->stream>>>(mul_arguments, count);
      goto done;
    }
    else if(l->precision<=512 && s->precision<=64) {
      configureActiveBlocks(handle, blocks, threads, regmp_mul_kernel<GSL, 16, 2>);
      regmp_mul_kernel<GSL, 16, 2><<<blocks, threads, 0, handle->stream>>>(mul_arguments, count);
      goto done;
    }
    else if(l->precision<=128 && s->precision<=128) {
      configureActiveBlocks(handle, blocks, threads, regmp_mul_kernel<GSL, 4, 4>);
      regmp_mul_kernel<GSL, 4, 4><<<blocks, threads, 0, handle->stream>>>(mul_arguments, count);
      goto done;
    }
    else if(l->precision<=256 && s->precision<=128) {
      configureActiveBlocks(handle, blocks, threads, regmp_mul_kernel<GSL, 8, 4>);
      regmp_mul_kernel<GSL, 8, 4><<<blocks, threads, 0, handle->stream>>>(mul_arguments, count);
      goto done;
    }
    else if(l->precision<=256 && s->precision<=256) {
      configureActiveBlocks(handle, blocks, threads, regmp_mul_kernel<GSL, 8, 8>);
      regmp_mul_kernel<GSL, 8, 8><<<blocks, threads, 0, handle->stream>>>(mul_arguments, count);
      goto done;
    }
    else if(l->precision<=384 && s->precision<=192) {
      configureActiveBlocks(handle, blocks, threads, regmp_mul_kernel<GSL, 12, 6>);
      regmp_mul_kernel<GSL, 12, 6><<<blocks, threads, 0, handle->stream>>>(mul_arguments, count);
      goto done;
    }
    else if(l->precision<=384 && s->precision<=384) {
      configureActiveBlocks(handle, blocks, threads, regmp_mul_kernel<GSL, 12, 12>);
      regmp_mul_kernel<GSL, 12, 12><<<blocks, threads, 0, handle->stream>>>(mul_arguments, count);
      goto done;
    }
    else if(l->precision<=512 && s->precision<=256) {
      configureActiveBlocks(handle, blocks, threads, regmp_mul_kernel<GSL, 16, 8>);
      regmp_mul_kernel<GSL, 16, 8><<<blocks, threads, 0, handle->stream>>>(mul_arguments, count);
      goto done;
    }
    else if(l->precision<=512 && s->precision<=512) {
      configureActiveBlocks(handle, blocks, threads, regmp_mul_kernel<GSL, 16, 16>);
      regmp_mul_kernel<GSL, 16, 16><<<blocks, threads, 0, handle->stream>>>(mul_arguments, count);
      goto done;
    }
  }
  else if(alg==xmpAlgorithmDigitMP) {
    configureActiveBlocks(handle, blocks, threads, digitmp_mul_kernel<GSL, DIGIT>);
    digitmp_mul_kernel<GSL, DIGIT><<<blocks, threads, 0, handle->stream>>>(mul_arguments, count);
    goto done;
  }

  //this is only reached if they requested an unsupported algorithm
  return xmpErrorUnsupported;
done:

  if(inplace)
    hipMemcpyAsync(p->slimbs,dst,out_size,hipMemcpyDeviceToDevice,handle->stream);

  p->setFormat(xmpFormatStrided);

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
//computes q=floor(a/b)
xmpError_t XMPAPI xmpIntegersDiv(xmpHandle_t handle, xmpIntegers_t q, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  xmpError_t error=xmpIntegersDivAsync(handle,q,a,b,count);
  if(error!=xmpErrorSuccess)
    return error;
  XMP_SET_DEVICE(handle);
  hipStreamSynchronize(handle->stream);
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
xmpError_t XMPAPI xmpIntegersDivAsync(xmpHandle_t handle, xmpIntegers_t q, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  dim3            blocks(DIV_ROUND_UP(count, GEOMETRY)), threads(GEOMETRY);
  int             device=handle->device;
  xmpError_t      error;
  div_arguments_t div_arguments;
  xmpExecutionPolicy_t policy=handle->policy;

  if(q->device!=device || a->device!=device || b->device!=device)
    return xmpErrorInvalidDevice;

  if(count==0)
    return xmpErrorSuccess;

  XMP_SET_DEVICE(handle);

  if(q->count<count)
    return xmpErrorInvalidCount;

  if(policy->indices[0] && policy->indices_count[0]<count)
    return xmpErrorInvalidCount;

  if(q->precision<a->precision)
    return xmpErrorInvalidPrecision;

  a->requireFormat(handle, xmpFormatStrided);
  b->requireFormat(handle, xmpFormatStrided);

  xmpAlgorithm_t alg = policy->algorithm;
  if(alg==xmpAlgorithmDefault) {
    if(a->precision<=512 && b->precision<512)
      alg = xmpAlgorithmRegMP;
    else
      alg = xmpAlgorithmDigitMP;
  }


  // package up the arguments
  div_arguments.out_data=q->slimbs;
  div_arguments.out_len=DIV_ROUND_UP(q->precision, 32);
  div_arguments.out_stride=q->stride;
  div_arguments.a_data=a->slimbs;
  div_arguments.a_len=DIV_ROUND_UP(a->precision, 32);
  div_arguments.a_stride=a->stride;
  div_arguments.a_count=a->count;
  div_arguments.b_data=b->slimbs;
  div_arguments.b_len=DIV_ROUND_UP(b->precision, 32);
  div_arguments.b_stride=b->stride;
  div_arguments.b_count=b->count;
  div_arguments.scratch=NULL;
  div_arguments.out_indices=policy->indices[0];
  div_arguments.a_indices=policy->indices[1];
  div_arguments.b_indices=policy->indices[2];
  div_arguments.a_indices_count=policy->indices_count[1];
  div_arguments.b_indices_count=policy->indices_count[2];

  //if q is in-place we need to work in scratch memory
  bool inplace= (q==a || q==b);
  size_t out_size=q->stride*q->nlimbs*sizeof(xmpLimb_t);

  if(inplace) {
    xmpError_t error;
    error=xmpSetNecessaryScratchSize(handle, out_size);
    if(error!=xmpErrorSuccess)
      return error;
    div_arguments.out_data=(xmpLimb_t*)handle->scratch;
  }

  if(alg==xmpAlgorithmRegMP) {
    if(a->precision<=64 && b->precision<=64) {
      configureActiveBlocks(handle, blocks, threads, regmp_div_kernel<GSL, 2, 2>);
      regmp_div_kernel<GSL, 2, 2><<<blocks, threads, 0, handle->stream>>>(div_arguments, count);
      goto done;
    }
    else if(a->precision<=128 && b->precision<=64) {
      configureActiveBlocks(handle, blocks, threads, regmp_div_kernel<GSL, 4, 2>);
      regmp_div_kernel<GSL, 4, 2><<<blocks, threads, 0, handle->stream>>>(div_arguments, count);
      goto done;
    }
    else if(a->precision<=128 && b->precision<=128) {
      configureActiveBlocks(handle, blocks, threads, regmp_div_kernel<GSL, 4, 4>);
      regmp_div_kernel<GSL, 4, 4><<<blocks, threads, 0, handle->stream>>>(div_arguments, count);
      goto done;
    }
    else if(a->precision<=256 && b->precision<=128) {
      configureActiveBlocks(handle, blocks, threads, regmp_div_kernel<GSL, 8, 4>);
      regmp_div_kernel<GSL, 8, 4><<<blocks, threads, 0, handle->stream>>>(div_arguments, count);
      goto done;
    }
    else if(a->precision<=256 && b->precision<=256) {
      configureActiveBlocks(handle, blocks, threads, regmp_div_kernel<GSL, 8, 8>);
      regmp_div_kernel<GSL, 8, 8><<<blocks, threads, 0, handle->stream>>>(div_arguments, count);
      goto done;
    }
    else if(a->precision<=384 && b->precision<=192) {
      configureActiveBlocks(handle, blocks, threads, regmp_div_kernel<GSL, 12, 6>);
      regmp_div_kernel<GSL, 12, 6><<<blocks, threads, 0, handle->stream>>>(div_arguments, count);
      goto done;
    }
    else if(a->precision<=384 && b->precision<=384) {
      configureActiveBlocks(handle, blocks, threads, regmp_div_kernel<GSL, 12, 12>);
      regmp_div_kernel<GSL, 12, 12><<<blocks, threads, 0, handle->stream>>>(div_arguments, count);
      goto done;
    }
    else if(a->precision<=512 && b->precision<=256) {
      configureActiveBlocks(handle, blocks, threads, regmp_div_kernel<GSL, 16, 8>);
      regmp_div_kernel<GSL, 16, 8><<<blocks, threads, 0, handle->stream>>>(div_arguments, count);
      goto done;
    }
    else if(a->precision<=512 && b->precision<=512) {
      configureActiveBlocks(handle, blocks, threads, regmp_div_kernel<GSL, 16, 16>);
      regmp_div_kernel<GSL, 16, 16><<<blocks, threads, 0, handle->stream>>>(div_arguments, count);
      goto done;
    }
  }
  else if(alg==xmpAlgorithmDigitMP) {
    int32_t digits=DIV_ROUND_UP(a->precision, DIGIT*32) + DIV_ROUND_UP(b->precision, DIGIT*32) + 2;
    size_t  bytes=digits*DIGIT*sizeof(xmpLimb_t);

    // FIX FIX FIX - need to set up scratch in all cases, not just digitized

    bytes=bytes*ROUND_UP(count, GEOMETRY);

    if(inplace) bytes+=out_size;

    error=xmpSetNecessaryScratchSize(handle, bytes);
    if(error!=xmpErrorSuccess)
      return error;

    if(inplace)
      div_arguments.scratch=(xmpLimb_t *)(reinterpret_cast<char*>(handle->scratch)+out_size);
    else
      div_arguments.scratch=(xmpLimb_t *)handle->scratch;

    configureActiveBlocks(handle, blocks, threads, digitmp_div_kernel<GSL, DIGIT>);
    digitmp_div_kernel<GSL, DIGIT><<<blocks, threads, 0, handle->stream>>>(div_arguments, count);
    goto done;
  }

  //this is only reached if they requested an unsupported algorithm
  return xmpErrorUnsupported;
done:
  q->setFormat(xmpFormatStrided);

  if(inplace) {
    hipMemcpyAsync(q->slimbs,div_arguments.out_data,out_size,hipMemcpyDeviceToDevice,handle->stream);
  }

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
//computes m=a%b
xmpError_t XMPAPI xmpIntegersMod(xmpHandle_t handle, xmpIntegers_t m, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  xmpError_t error=xmpIntegersModAsync(handle,m,a,b,count);
  if(error!=xmpErrorSuccess)
    return error;
  XMP_SET_DEVICE(handle);
  hipStreamSynchronize(handle->stream);
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
xmpError_t XMPAPI xmpIntegersModAsync(xmpHandle_t handle, xmpIntegers_t m, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  dim3            blocks(DIV_ROUND_UP(count, GEOMETRY)), threads(GEOMETRY);
  int             device=handle->device;
  xmpError_t      error;
  mod_arguments_t mod_arguments;
  xmpExecutionPolicy_t policy=handle->policy;

  if(m->device!=device || a->device!=device || b->device!=device)
    return xmpErrorInvalidDevice;

  if(count==0)
    return xmpErrorSuccess;

  XMP_SET_DEVICE(handle);

  if(m->count<count)
    return xmpErrorInvalidCount;

  if(policy->indices[0] && policy->indices_count[0]<count)
    return xmpErrorInvalidCount;

  if(m->precision<b->precision)
    return xmpErrorInvalidPrecision;

  a->requireFormat(handle, xmpFormatStrided);
  b->requireFormat(handle, xmpFormatStrided);

  xmpAlgorithm_t alg = policy->algorithm;
  if(alg==xmpAlgorithmDefault) {
    if(a->precision<=512 && b->precision<512)
      alg = xmpAlgorithmRegMP;
    else
      alg = xmpAlgorithmDigitMP;
  }

  // package up the arguments
  mod_arguments.out_data=m->slimbs;
  mod_arguments.out_len=DIV_ROUND_UP(m->precision, 32);
  mod_arguments.out_stride=m->stride;
  mod_arguments.a_data=a->slimbs;
  mod_arguments.a_len=DIV_ROUND_UP(a->precision, 32);
  mod_arguments.a_stride=a->stride;
  mod_arguments.a_count=a->count;
  mod_arguments.b_data=b->slimbs;
  mod_arguments.b_len=DIV_ROUND_UP(b->precision, 32);
  mod_arguments.b_stride=b->stride;
  mod_arguments.b_count=b->count;
  mod_arguments.scratch=NULL;
  mod_arguments.out_indices=policy->indices[0];
  mod_arguments.a_indices=policy->indices[1];
  mod_arguments.b_indices=policy->indices[2];
  mod_arguments.a_indices_count=policy->indices_count[1];
  mod_arguments.b_indices_count=policy->indices_count[2];

  //if m is in-place we need to work in scratch memory
  bool inplace=(m==a || m==b);
  size_t out_size=m->stride*m->nlimbs*sizeof(xmpLimb_t);

  if(inplace) {
    xmpError_t error;
    error=xmpSetNecessaryScratchSize(handle, out_size);
    if(error!=xmpErrorSuccess)
      return error;
    mod_arguments.out_data=(xmpLimb_t*)handle->scratch;
  }

  if(alg==xmpAlgorithmRegMP) {
    if(a->precision<=64 && b->precision<=64) {
      configureActiveBlocks(handle, blocks, threads, regmp_mod_kernel<GSL, 2, 2>);
      regmp_mod_kernel<GSL, 2, 2><<<blocks, threads, 0, handle->stream>>>(mod_arguments, count);
      goto done;
    }
    else if(a->precision<=128 && b->precision<=64) {
      configureActiveBlocks(handle, blocks, threads, regmp_mod_kernel<GSL, 4, 2>);
      regmp_mod_kernel<GSL, 4, 2><<<blocks, threads, 0, handle->stream>>>(mod_arguments, count);
      goto done;
    }
    else if(a->precision<=128 && b->precision<=128) {
      configureActiveBlocks(handle, blocks, threads, regmp_mod_kernel<GSL, 4, 4>);
      regmp_mod_kernel<GSL, 4, 4><<<blocks, threads, 0, handle->stream>>>(mod_arguments, count);
      goto done;
    }
    else if(a->precision<=256 && b->precision<=128) {
      configureActiveBlocks(handle, blocks, threads, regmp_mod_kernel<GSL, 8, 4>);
      regmp_mod_kernel<GSL, 8, 4><<<blocks, threads, 0, handle->stream>>>(mod_arguments, count);
      goto done;
    }
    else if(a->precision<=256 && b->precision<=256) {
      configureActiveBlocks(handle, blocks, threads, regmp_mod_kernel<GSL, 8, 8>);
      regmp_mod_kernel<GSL, 8, 8><<<blocks, threads, 0, handle->stream>>>(mod_arguments, count);
      goto done;
    }
    else if(a->precision<=384 && b->precision<=192) {
      configureActiveBlocks(handle, blocks, threads, regmp_mod_kernel<GSL, 12, 6>);
      regmp_mod_kernel<GSL, 12, 6><<<blocks, threads, 0, handle->stream>>>(mod_arguments, count);
      goto done;
    }
    else if(a->precision<=384 && b->precision<=384) {
      configureActiveBlocks(handle, blocks, threads, regmp_mod_kernel<GSL, 12, 12>);
      regmp_mod_kernel<GSL, 12, 12><<<blocks, threads, 0, handle->stream>>>(mod_arguments, count);
      goto done;
    }
    else if(a->precision<=512 && b->precision<=256) {
      configureActiveBlocks(handle, blocks, threads, regmp_mod_kernel<GSL, 16, 8>);
      regmp_mod_kernel<GSL, 16, 8><<<blocks, threads, 0, handle->stream>>>(mod_arguments, count);
      goto done;
    }
    else if(a->precision<=512 && b->precision<=512) {
      configureActiveBlocks(handle, blocks, threads, regmp_mod_kernel<GSL, 16, 16>);
      regmp_mod_kernel<GSL, 16, 16><<<blocks, threads, 0, handle->stream>>>(mod_arguments, count);
      goto done;
    }
  }
  else if(alg==xmpAlgorithmDigitMP) {
    int32_t digits=DIV_ROUND_UP(a->precision, DIGIT*32) + DIV_ROUND_UP(b->precision, DIGIT*32) + 2;
    size_t  bytes=digits*DIGIT*sizeof(xmpLimb_t);

    bytes=bytes*ROUND_UP(count, GEOMETRY);

    if(inplace) bytes+=out_size;

    error=xmpSetNecessaryScratchSize(handle, bytes);
    if(error!=xmpErrorSuccess)
      return error;

    if(inplace)
      mod_arguments.scratch=(xmpLimb_t *)(reinterpret_cast<char*>(handle->scratch)+out_size);
    else
      mod_arguments.scratch=(xmpLimb_t *)handle->scratch;

    configureActiveBlocks(handle, blocks, threads, digitmp_mod_kernel<GSL, DIGIT>);
    digitmp_mod_kernel<GSL, DIGIT><<<blocks, threads, 0, handle->stream>>>(mod_arguments, count);
    goto done;
  }

  //this is only reached if they requested an unsupported algorithm
  return xmpErrorUnsupported;
done:

  if(inplace) {
    hipMemcpyAsync(m->slimbs,mod_arguments.out_data,out_size,hipMemcpyDeviceToDevice,handle->stream);
  }

  m->setFormat(xmpFormatStrided);

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
//computes q=floor(a/b) and m=a%b
xmpError_t XMPAPI xmpIntegersDivMod(xmpHandle_t handle, xmpIntegers_t q, xmpIntegers_t m, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  xmpError_t error=xmpIntegersDivModAsync(handle,q,m,a,b,count);
  if(error!=xmpErrorSuccess)
    return error;
  XMP_SET_DEVICE(handle);
  hipStreamSynchronize(handle->stream);
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
xmpError_t XMPAPI xmpIntegersDivModAsync(xmpHandle_t handle, xmpIntegers_t q, xmpIntegers_t m, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  dim3               blocks(DIV_ROUND_UP(count, GEOMETRY)), threads(GEOMETRY);
  int                device=handle->device;
  xmpError_t         error;
  divmod_arguments_t divmod_arguments;
  xmpExecutionPolicy_t policy=handle->policy;

  if(q->device!=device || m->device!=device || a->device!=device || b->device!=device)
    return xmpErrorInvalidDevice;

  if(count==0)
    return xmpErrorSuccess;

  XMP_SET_DEVICE(handle);

  if(m->count<count || q->count<count)
    return xmpErrorInvalidCount;

  if(policy->indices[0] && policy->indices_count[0]<count)
    return xmpErrorInvalidCount;

  if(policy->indices[1] && policy->indices_count[1]<count)
    return xmpErrorInvalidCount;

  if(q->precision<a->precision || m->precision<b->precision)
    return xmpErrorInvalidPrecision;

  a->requireFormat(handle, xmpFormatStrided);
  b->requireFormat(handle, xmpFormatStrided);

  xmpAlgorithm_t alg = policy->algorithm;
  if(alg==xmpAlgorithmDefault) {
    if(a->precision<=512 && b->precision<512)
      alg = xmpAlgorithmRegMP;
    else
      alg = xmpAlgorithmDigitMP;
  }

  // package up the arguments
  divmod_arguments.q_data=q->slimbs;
  divmod_arguments.q_len=DIV_ROUND_UP(q->precision, 32);
  divmod_arguments.q_stride=q->stride;
  divmod_arguments.m_data=m->slimbs;
  divmod_arguments.m_len=DIV_ROUND_UP(m->precision, 32);
  divmod_arguments.m_stride=m->stride;
  divmod_arguments.a_data=a->slimbs;
  divmod_arguments.a_len=DIV_ROUND_UP(a->precision, 32);
  divmod_arguments.a_stride=a->stride;
  divmod_arguments.a_count=a->count;
  divmod_arguments.b_data=b->slimbs;
  divmod_arguments.b_len=DIV_ROUND_UP(b->precision, 32);
  divmod_arguments.b_stride=b->stride;
  divmod_arguments.b_count=b->count;
  divmod_arguments.scratch=NULL;
  divmod_arguments.q_indices=policy->indices[0];
  divmod_arguments.r_indices=policy->indices[1];
  divmod_arguments.a_indices=policy->indices[2];
  divmod_arguments.b_indices=policy->indices[3];
  divmod_arguments.a_indices_count=policy->indices_count[2];
  divmod_arguments.b_indices_count=policy->indices_count[3];

  //if q or m is in-place we need to work in scratch memory
  bool qinplace=(q==a || q==b);
  bool minplace=(m==a || m==b);
  size_t qout_size=q->stride*q->nlimbs*sizeof(xmpLimb_t);
  size_t mout_size=m->stride*m->nlimbs*sizeof(xmpLimb_t);

  int num_inplace = qinplace+minplace;

  if(num_inplace>0) {
    xmpError_t error;
    error=xmpSetNecessaryScratchSize(handle, qout_size+mout_size);
    if(error!=xmpErrorSuccess)
      return error;
  }
  size_t soffset=0;
  if(qinplace) {
    divmod_arguments.q_indices=(xmpLimb_t*)handle->scratch;
    soffset+=qout_size;
  }
  if(minplace) {
    divmod_arguments.r_indices=(xmpLimb_t*)(reinterpret_cast<char*>(handle->scratch)+soffset);
  }

  if(alg==xmpAlgorithmRegMP) {
    if(a->precision<=64 && b->precision<=64) {
      configureActiveBlocks(handle, blocks, threads, regmp_divmod_kernel<GSL, 2, 2>);
      regmp_divmod_kernel<GSL, 2, 2><<<blocks, threads, 0, handle->stream>>>(divmod_arguments, count);
      goto done;
    }
    else if(a->precision<=128 && b->precision<=64) {
      configureActiveBlocks(handle, blocks, threads, regmp_divmod_kernel<GSL, 4, 2>);
      regmp_divmod_kernel<GSL, 4, 2><<<blocks, threads, 0, handle->stream>>>(divmod_arguments, count);
      goto done;
    }
    else if(a->precision<=128 && b->precision<=128) {
      configureActiveBlocks(handle, blocks, threads, regmp_divmod_kernel<GSL, 4, 4>);
      regmp_divmod_kernel<GSL, 4, 4><<<blocks, threads, 0, handle->stream>>>(divmod_arguments, count);
      goto done;
    }
    else if(a->precision<=256 && b->precision<=128) {
      configureActiveBlocks(handle, blocks, threads, regmp_divmod_kernel<GSL, 8, 4>);
      regmp_divmod_kernel<GSL, 8, 4><<<blocks, threads, 0, handle->stream>>>(divmod_arguments, count);
      goto done;
    }
    else if(a->precision<=256 && b->precision<=256) {
      configureActiveBlocks(handle, blocks, threads, regmp_divmod_kernel<GSL, 8, 8>);
      regmp_divmod_kernel<GSL, 8, 8><<<blocks, threads, 0, handle->stream>>>(divmod_arguments, count);
      goto done;
    }
    else if(a->precision<=384 && b->precision<=192) {
      configureActiveBlocks(handle, blocks, threads, regmp_divmod_kernel<GSL, 12, 6>);
      regmp_divmod_kernel<GSL, 12, 6><<<blocks, threads, 0, handle->stream>>>(divmod_arguments, count);
      goto done;
    }
    else if(a->precision<=384 && b->precision<=384) {
      configureActiveBlocks(handle, blocks, threads, regmp_divmod_kernel<GSL, 12, 12>);
      regmp_divmod_kernel<GSL, 12, 12><<<blocks, threads, 0, handle->stream>>>(divmod_arguments, count);
      goto done;
    }
    else if(a->precision<=512 && b->precision<=256) {
      configureActiveBlocks(handle, blocks, threads, regmp_divmod_kernel<GSL, 16, 8>);
      regmp_divmod_kernel<GSL, 16, 8><<<blocks, threads, 0, handle->stream>>>(divmod_arguments, count);
      goto done;
    }
    else if(a->precision<=512 && b->precision<=512) {
      configureActiveBlocks(handle, blocks, threads, regmp_divmod_kernel<GSL, 16, 16>);
      regmp_divmod_kernel<GSL, 16, 16><<<blocks, threads, 0, handle->stream>>>(divmod_arguments, count);
      goto done;
    }
  }
  else if(alg==xmpAlgorithmDigitMP) {
    int32_t digits=DIV_ROUND_UP(a->precision, DIGIT*32) + DIV_ROUND_UP(b->precision, DIGIT*32) + 2;
    size_t  bytes=digits*DIGIT*sizeof(xmpLimb_t);

    bytes=bytes*ROUND_UP(count, GEOMETRY);

    error=xmpSetNecessaryScratchSize(handle, bytes);
    if(error!=xmpErrorSuccess)
      return error;

    divmod_arguments.scratch=(xmpLimb_t *)handle->scratch;

    configureActiveBlocks(handle, blocks, threads, digitmp_divmod_kernel<GSL, DIGIT>);
    digitmp_divmod_kernel<GSL, DIGIT><<<blocks, threads, 0, handle->stream>>>(divmod_arguments, count);
    goto done;
  }

  //this is only reached if they requested an unsupported algorithm
  return xmpErrorUnsupported;
done:

  if(qinplace) {
    hipMemcpyAsync(q->slimbs,divmod_arguments.q_indices,qout_size,hipMemcpyDeviceToDevice,handle->stream);
  }
  if(minplace) {
    hipMemcpyAsync(m->slimbs,divmod_arguments.r_indices,mout_size,hipMemcpyDeviceToDevice,handle->stream);
  }

  q->setFormat(xmpFormatStrided);
  m->setFormat(xmpFormatStrided);

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

//computes out=base^exp % mod for count integers
xmpError_t XMPAPI xmpIntegersPowm(xmpHandle_t handle, xmpIntegers_t out, const xmpIntegers_t a, const xmpIntegers_t exp, const xmpIntegers_t mod, uint32_t count) {
  xmpError_t error=xmpIntegersPowmAsync(handle,out,a,exp,mod,count);
  if(error!=xmpErrorSuccess)
    return error;
  XMP_SET_DEVICE(handle);
  hipStreamSynchronize(handle->stream);
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

xmpError_t XMPAPI xmpIntegersPowmAsync(xmpHandle_t handle, xmpIntegers_t out, const xmpIntegers_t a, const xmpIntegers_t exp, const xmpIntegers_t mod, uint32_t count) {
  int                  device=handle->device;
  xmpExecutionPolicy_t policy=handle->policy;
  //verify out, base, exp, mod devices all match handle device
  if(out->device!=device || a->device!=device || exp->device!=device || mod->device!=device)
    return xmpErrorInvalidDevice;

  if(count==0)
    return xmpErrorSuccess;

  int32_t precision=out->precision;

  if(out->count<count)
    return xmpErrorInvalidCount;

  if(policy->indices[0] && policy->indices_count[0]<count)
    return xmpErrorInvalidCount;

  if(out->precision!=precision || a->precision!=precision || mod->precision!=precision)
    return xmpErrorInvalidPrecision;

  xmpAlgorithm_t alg = policy->algorithm;
#if 1
  uint32_t start=0;
  while(start<count) {
    LaunchParameters params=getPowmLaunchParameters(handle,precision,count-start,alg);
    if(params.alg_index==-1)
      return xmpErrorUnsupported;
    uint32_t lcount=params.count;
    xmpPowmAlgorithm algorithm=xmpPowmAlgorithms[params.alg_index];
    xmpError_t error=algorithm.pfunc(handle, out, a, exp, mod, start, lcount, NULL, NULL);
    if(error!=xmpErrorSuccess) {
      return error;
    }
    start+=lcount;
  }
  return xmpErrorSuccess;
#else
  if(alg==xmpAlgorithmDefault) {
    if(count<handle->smCount*512 && handle->arch>=30 && precision<=8192)
      alg=xmpAlgorithmDistributedMP;   // for a small number of instances, use distributed
    else if(precision<=512)
      alg=xmpAlgorithmRegMP;
    else if(precision<=8192)
      alg=xmpAlgorithmDistributedMP;   // for now... distributed seems to always outperfm digitized
    else
      alg=xmpAlgorithmDigitMP;
  }

  if(alg==xmpAlgorithmRegMP) {
    out->setFormat(xmpFormatStrided);
    if(precision<=128)
      return internalPowmRegMP<128, 4, 4, 0, 0>(handle, out, a, exp, mod, 0, count, NULL, NULL);
    else if(precision<=256)
      return internalPowmRegMP<128, 4, 8, 0, 0>(handle, out, a, exp, mod, 0, count, NULL, NULL);
    else if(precision<=384)
      return internalPowmRegMP<128, 4, 12, 0, 0>(handle, out, a, exp, mod, 0, count, NULL, NULL);
    else if(precision<=512)
      return internalPowmRegMP<128, 4, 16, 0, 0>(handle, out, a, exp, mod, 0, count, NULL, NULL);
    else
      return xmpErrorUnsupported;
  }

  if(alg==xmpAlgorithmDigitMP) {
    out->setFormat(xmpFormatStrided);
    return internalPowmDigitMP<128, 4, DIGIT>(handle, out, a, exp, mod, 0, count, NULL, NULL);
  }

  if(alg==xmpAlgorithmDistributedMP) {
    out->setFormat(xmpFormatCompact);   // so I can test before the copy out kernels are done
    if(count<handle->smCount*256) {
      // use smallest number of words, to achieve lowest latency
      if(precision<=128)
        return internalPowmWarpDistributedMP<128, 4, 4, 1>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=256)
        return internalPowmWarpDistributedMP<128, 4, 8, 1>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=384)
        return internalPowmWarpDistributedMP<128, 4, 4, 3>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=512)
        return internalPowmWarpDistributedMP<128, 4, 16, 1>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=1024)
        return internalPowmWarpDistributedMP<128, 4, 32, 1>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=1536)
        return internalPowmWarpDistributedMP<128, 4, 16, 3>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=2048)
        return internalPowmWarpDistributedMP<128, 4, 32, 2>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=3072)
        return internalPowmWarpDistributedMP<128, 4, 32, 3>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=4096)
        return internalPowmWarpDistributedMP<128, 4, 32, 4>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=6144)
        return internalPowmWarpDistributedMP<128, 4, 32, 6>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=8192)
        return internalPowmWarpDistributedMP<128, 4, 32, 8>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else
        return xmpErrorUnsupported;
    }
    else {
      // use largest number of words to achieve highest throughput
      if(precision<=128)
        return internalPowmWarpDistributedMP<128, 4, 2, 2>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=256)
        return internalPowmWarpDistributedMP<128, 4, 2, 4>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=384)
        return internalPowmWarpDistributedMP<128, 4, 2, 6>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=512)
        return internalPowmWarpDistributedMP<128, 4, 2, 8>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=1024)
        return internalPowmWarpDistributedMP<128, 4, 4, 8>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=1536)
        return internalPowmWarpDistributedMP<128, 4, 8, 6>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=2048)
        return internalPowmWarpDistributedMP<128, 4, 8, 8>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=3072)
        return internalPowmWarpDistributedMP<128, 4, 16, 6>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=4096)
        return internalPowmWarpDistributedMP<128, 4, 16, 8>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=6144)
        return internalPowmWarpDistributedMP<128, 4, 32, 6>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else if(precision<=8192)
        return internalPowmWarpDistributedMP<128, 4, 32, 8>(handle, out, a, exp, mod, 0, count, NULL, NULL);
      else
        return xmpErrorUnsupported;
    }
  }

  return xmpErrorUnsupported;
#endif
}

//compute c=CMP(a,b),  -1 a is smaller, 0 equal, +1 a is larger
xmpError_t XMPAPI xmpIntegersCmp(xmpHandle_t handle, int32_t *c, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  xmpError_t error=xmpIntegersCmpAsync(handle,c,a,b,count);
  if(error!=xmpErrorSuccess)
    return error;
  XMP_SET_DEVICE(handle);
  hipStreamSynchronize(handle->stream);
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
xmpError_t XMPAPI xmpIntegersCmpAsync(xmpHandle_t handle, int32_t *c, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  dim3            blocks(DIV_ROUND_UP(count, GEOMETRY)), threads(GEOMETRY);
  int             device=handle->device;
  int32_t         negate;
  xmpIntegers_t   l, s;
  xmpError_t      error;
  cmp_arguments_t cmp_arguments;
  xmpExecutionPolicy_t policy=handle->policy;

  if(a->device!=device || b->device!=device)
    return xmpErrorInvalidDevice;

  if(count==0)
    return xmpErrorSuccess;

  XMP_SET_DEVICE(handle);

  error=xmpSetNecessaryScratchSize(handle, count*sizeof(int32_t));
  if(error!=xmpErrorSuccess)
    return error;

  if(a->precision>=b->precision) {
    negate=1;
    l=a;
    s=b;
  }
  else {
    negate=-1;
    l=b;
    s=a;
  }

  //check if we know where this pointer came from, if not assume host
  hipPointerAttribute_t attrib;
  hipError_t cerror=hipPointerGetAttributes(&attrib,c);
  if(cerror!=hipSuccess) {
    if(cerror==hipErrorInvalidValue) {
      hipGetLastError();  //reset to hipSuccess
      attrib.memoryType=hipMemoryTypeHost;
    }
    else {
      return xmpErrorCuda;
    }
  }

  int32_t *dst=c;
  if(attrib.memoryType==hipMemoryTypeHost) {
    xmpError_t error=xmpSetNecessaryScratchSize(handle, count*sizeof(int32_t));
    if(error!=xmpErrorSuccess)
      return error;

    dst=(int32_t*)handle->scratch;
  }

  l->requireFormat(handle, xmpFormatStrided);
  s->requireFormat(handle, xmpFormatStrided);

  cmp_arguments.out_data=dst;
  cmp_arguments.a_data=l->slimbs;
  cmp_arguments.b_data=s->slimbs;
  cmp_arguments.a_len=DIV_ROUND_UP(l->precision, 32);
  cmp_arguments.a_stride=l->stride;
  cmp_arguments.a_count=l->count;
  cmp_arguments.b_len=DIV_ROUND_UP(s->precision, 32);
  cmp_arguments.b_stride=s->stride;
  cmp_arguments.b_count=s->count;
  cmp_arguments.negate=negate;
  cmp_arguments.a_indices=policy->indices[0];
  cmp_arguments.b_indices=policy->indices[1];
  cmp_arguments.a_indices_count=policy->indices_count[0];
  cmp_arguments.b_indices_count=policy->indices_count[1];

  configureActiveBlocks(handle, blocks, threads, strided_compare_kernel<GSL>);
  strided_compare_kernel<GSL><<<blocks, threads, 0, handle->stream>>>(cmp_arguments, count);

  if(attrib.memoryType==hipMemoryTypeHost) {
    hipMemcpyAsync(c,dst,sizeof(int32_t)*count,hipMemcpyDefault,handle->stream);
  }
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}

//computes c=shift(a,shift)
xmpError_t XMPAPI xmpIntegersShf(xmpHandle_t handle, xmpIntegers_t c, const xmpIntegers_t a, const int32_t* shift, const uint32_t shift_count, uint32_t count) {
  xmpError_t error=xmpIntegersShfAsync(handle,c,a,shift,shift_count,count);
  if(error!=xmpErrorSuccess)
    return error;
  XMP_SET_DEVICE(handle);
  hipStreamSynchronize(handle->stream);
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
xmpError_t XMPAPI xmpIntegersShfAsync(xmpHandle_t handle, xmpIntegers_t c, const xmpIntegers_t a, const int32_t* shift, const uint32_t shift_count, uint32_t count) {
  dim3            blocks(DIV_ROUND_UP(count, GEOMETRY)), threads(GEOMETRY);
  int             device=handle->device;
  shf_arguments_t shf_arguments;
  xmpExecutionPolicy_t policy=handle->policy;

  XMP_CHECK_NE(shift,NULL);

  if(c->device!=device || a->device!=device)
    return xmpErrorInvalidDevice;

  if(count==0)
    return xmpErrorSuccess;

  XMP_SET_DEVICE(handle);

  if(c->count<count)
    return xmpErrorInvalidCount;

  if(policy->indices[0] && policy->indices_count[0]<count)
    return xmpErrorInvalidCount;

  a->requireFormat(handle, xmpFormatStrided);

  //check if we know where this pointer came from, if not assume host
  hipPointerAttribute_t attrib;
  hipError_t error=hipPointerGetAttributes(&attrib,shift);
  if(error!=hipSuccess) {
    if(error==hipErrorInvalidValue) {
      hipGetLastError();  //reset to hipSuccess
      attrib.memoryType=hipMemoryTypeHost;
    } else {
      return xmpErrorCuda;
    }
  }

  uint32_t scount=MIN(shift_count,count);
  int32_t *src=const_cast<int32_t*>(shift);

  size_t scratch_size_out=0, scratch_size_shift=0;
  if(a==c)
    scratch_size_out=a->stride*a->nlimbs*sizeof(xmpLimb_t);
  if(attrib.memoryType==hipMemoryTypeHost)
    scratch_size_shift=scount*sizeof(int32_t);

  xmpError_t xerror=xmpSetNecessaryScratchSize(handle, scratch_size_out+scratch_size_shift);
  if(xerror!=xmpErrorSuccess)
    return xerror;

  xmpLimb_t *dst=  (a==c) ? (xmpLimb_t*) handle->scratch : c->slimbs;

  if(attrib.memoryType==hipMemoryTypeHost) {
    src=(int32_t*)((char*)handle->scratch+scratch_size_out);
    hipMemcpyAsync(src,shift,scratch_size_shift,hipMemcpyHostToDevice,handle->stream);
  }

  shf_arguments.out_data=(xmpLimb_t *)dst;
  shf_arguments.a_data=a->slimbs;
  shf_arguments.shift_data=(int32_t *)src;
  shf_arguments.out_len=DIV_ROUND_UP(c->precision, 32);
  shf_arguments.out_stride=c->stride;
  shf_arguments.a_len=DIV_ROUND_UP(a->precision, 32);
  shf_arguments.a_stride=a->stride;
  shf_arguments.a_count=a->count;
  shf_arguments.shift_count=scount;
  shf_arguments.out_indices=policy->indices[0];
  shf_arguments.a_indices=policy->indices[1];
  shf_arguments.a_indices_count=policy->indices_count[1];

  configureActiveBlocks(handle, blocks, threads, strided_shf_kernel<GSL>);
  strided_shf_kernel<GSL><<<blocks, threads, 0, handle->stream>>>(shf_arguments, count);

  if(a==c) {
    hipMemcpyAsync(c->slimbs,dst,scratch_size_out,hipMemcpyDeviceToDevice,handle->stream);
  }

  c->setFormat(xmpFormatStrided);

  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}


//computes c=a|b
xmpError_t XMPAPI xmpIntegersIor(xmpHandle_t handle, xmpIntegers_t c, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  xmpError_t error=xmpIntegersIorAsync(handle,c,a,b,count);
  if(error!=xmpErrorSuccess)
    return error;
  XMP_SET_DEVICE(handle);
  hipStreamSynchronize(handle->stream);
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
xmpError_t XMPAPI xmpIntegersIorAsync(xmpHandle_t handle, xmpIntegers_t c, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  dim3            blocks(DIV_ROUND_UP(count, GEOMETRY)), threads(GEOMETRY);
  int             device=handle->device;
  xmpIntegers_t   l, s;
  ior_arguments_t ior_arguments;
  xmpExecutionPolicy_t policy=handle->policy;

  if(c->device!=device || a->device!=device || b->device!=device)
    return xmpErrorInvalidDevice;

  if(count==0)
    return xmpErrorSuccess;

  XMP_SET_DEVICE(handle);

  if(c->count<count)
    return xmpErrorInvalidCount;

  if(policy->indices[0] && policy->indices_count[0]<count)
    return xmpErrorInvalidCount;

  if(c->precision<MAX(a->precision, b->precision))
    return xmpErrorInvalidPrecision;

  if(a->precision>=b->precision) {
    l=a;
    s=b;
  }
  else {
    l=b;
    s=a;
  }

  l->requireFormat(handle, xmpFormatStrided);
  s->requireFormat(handle, xmpFormatStrided);

  // package up the arguments
  ior_arguments.out_data=c->slimbs;
  ior_arguments.out_len=DIV_ROUND_UP(c->precision, 32);
  ior_arguments.out_stride=c->stride;
  ior_arguments.a_data=l->slimbs;
  ior_arguments.a_len=DIV_ROUND_UP(l->precision, 32);
  ior_arguments.a_stride=l->stride;
  ior_arguments.a_count=l->count;
  ior_arguments.b_data=s->slimbs;
  ior_arguments.b_len=DIV_ROUND_UP(s->precision, 32);
  ior_arguments.b_stride=s->stride;
  ior_arguments.b_count=s->count;
  ior_arguments.out_indices=policy->indices[0];
  ior_arguments.a_indices=policy->indices[1];
  ior_arguments.b_indices=policy->indices[2];
  ior_arguments.a_indices_count=policy->indices_count[1];
  ior_arguments.b_indices_count=policy->indices_count[2];


  //if c is indexed and in-place we need to work in scratch memory
  bool inplace=(c==a || c==b);
  size_t out_size=c->stride*c->nlimbs*sizeof(xmpLimb_t);

  if(inplace) {
    xmpError_t error;
    error=xmpSetNecessaryScratchSize(handle, out_size);
    if(error!=xmpErrorSuccess)
      return error;
    ior_arguments.out_data=(xmpLimb_t*)handle->scratch;
  }

  configureActiveBlocks(handle, blocks, threads, strided_ior_kernel<GSL>);
  strided_ior_kernel<GSL><<<blocks, threads, 0, handle->stream>>>(ior_arguments, count);


  if(inplace) {
    hipMemcpyAsync(c->slimbs,ior_arguments.out_data,out_size,hipMemcpyDeviceToDevice,handle->stream);
  }

  c->setFormat(xmpFormatStrided);
  return xmpErrorSuccess;
}
//computes c=a&b
xmpError_t XMPAPI xmpIntegersAnd(xmpHandle_t handle, xmpIntegers_t c, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  xmpError_t error=xmpIntegersAndAsync(handle,c,a,b,count);
  if(error!=xmpErrorSuccess)
    return error;
  XMP_SET_DEVICE(handle);
  hipStreamSynchronize(handle->stream);
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
xmpError_t XMPAPI xmpIntegersAndAsync(xmpHandle_t handle, xmpIntegers_t c, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  dim3            blocks(DIV_ROUND_UP(count, GEOMETRY)), threads(GEOMETRY);
  int             device=handle->device;
  xmpIntegers_t   l, s;
  and_arguments_t and_arguments;
  xmpExecutionPolicy_t policy=handle->policy;

  if(c->device!=device || a->device!=device || b->device!=device)
    return xmpErrorInvalidDevice;

  if(count==0)
    return xmpErrorSuccess;

  XMP_SET_DEVICE(handle);

  if(c->count<count)
    return xmpErrorInvalidCount;

  if(policy->indices[0] && policy->indices_count[0]<count)
    return xmpErrorInvalidCount;

  if(c->precision<MAX(a->precision, b->precision))
    return xmpErrorInvalidPrecision;

  if(a->precision>=b->precision) {
    l=a;
    s=b;
  }
  else {
    l=b;
    s=a;
  }

  l->requireFormat(handle, xmpFormatStrided);
  s->requireFormat(handle, xmpFormatStrided);

  // package up the arguments
  and_arguments.out_data=c->slimbs;
  and_arguments.out_len=DIV_ROUND_UP(c->precision, 32);
  and_arguments.out_stride=c->stride;
  and_arguments.a_data=l->slimbs;
  and_arguments.a_len=DIV_ROUND_UP(l->precision, 32);
  and_arguments.a_stride=l->stride;
  and_arguments.a_count=l->count;
  and_arguments.b_data=s->slimbs;
  and_arguments.b_len=DIV_ROUND_UP(s->precision, 32);
  and_arguments.b_stride=s->stride;
  and_arguments.b_count=s->count;
  and_arguments.out_indices=policy->indices[0];
  and_arguments.a_indices=policy->indices[1];
  and_arguments.b_indices=policy->indices[2];
  and_arguments.a_indices_count=policy->indices_count[1];
  and_arguments.b_indices_count=policy->indices_count[2];

  //if c is indexed and in-place we need to work in scratch memory
  bool inplace=(c==a || c==b);
  size_t out_size=c->stride*c->nlimbs*sizeof(xmpLimb_t);

  if(inplace) {
    xmpError_t error;
    error=xmpSetNecessaryScratchSize(handle, out_size);
    if(error!=xmpErrorSuccess)
      return error;
    and_arguments.out_data=(xmpLimb_t*)handle->scratch;
  }

  configureActiveBlocks(handle, blocks, threads, strided_and_kernel<GSL>);
  strided_and_kernel<GSL><<<blocks, threads, 0, handle->stream>>>(and_arguments, count);

  if(inplace) {
    hipMemcpyAsync(c->slimbs,and_arguments.out_data,out_size,hipMemcpyDeviceToDevice,handle->stream);
  }

  c->setFormat(xmpFormatStrided);
  return xmpErrorSuccess;
}
//computes c=a^b
xmpError_t XMPAPI xmpIntegersXor(xmpHandle_t handle, xmpIntegers_t c, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  xmpError_t error=xmpIntegersXorAsync(handle,c,a,b,count);
  if(error!=xmpErrorSuccess)
    return error;
  XMP_SET_DEVICE(handle);
  hipStreamSynchronize(handle->stream);
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
xmpError_t XMPAPI xmpIntegersXorAsync(xmpHandle_t handle, xmpIntegers_t c, const xmpIntegers_t a, const xmpIntegers_t b, uint32_t count) {
  dim3            blocks(DIV_ROUND_UP(count, GEOMETRY)), threads(GEOMETRY);
  int             device=handle->device;
  xmpIntegers_t   l, s;
  xor_arguments_t xor_arguments;
  xmpExecutionPolicy_t policy=handle->policy;

  if(c->device!=device || a->device!=device || b->device!=device)
    return xmpErrorInvalidDevice;

  if(count==0)
    return xmpErrorSuccess;

  XMP_SET_DEVICE(handle);

  if(c->count<count)
    return xmpErrorInvalidCount;

  if(policy->indices[0] && policy->indices_count[0]<count)
    return xmpErrorInvalidCount;

  if(c->precision<MAX(a->precision, b->precision))
    return xmpErrorInvalidPrecision;

  if(a->precision>=b->precision) {
    l=a;
    s=b;
  }
  else {
    l=b;
    s=a;
  }

  l->requireFormat(handle, xmpFormatStrided);
  s->requireFormat(handle, xmpFormatStrided);

  // package up the arguments
  xor_arguments.out_data=c->slimbs;
  xor_arguments.out_len=DIV_ROUND_UP(c->precision, 32);
  xor_arguments.out_stride=c->stride;
  xor_arguments.a_data=l->slimbs;
  xor_arguments.a_len=DIV_ROUND_UP(l->precision, 32);
  xor_arguments.a_stride=l->stride;
  xor_arguments.a_count=l->count;
  xor_arguments.b_data=s->slimbs;
  xor_arguments.b_len=DIV_ROUND_UP(s->precision, 32);
  xor_arguments.b_stride=s->stride;
  xor_arguments.b_count=s->count;
  xor_arguments.out_indices=policy->indices[0];
  xor_arguments.a_indices=policy->indices[1];
  xor_arguments.b_indices=policy->indices[2];
  xor_arguments.a_indices_count=policy->indices_count[1];
  xor_arguments.b_indices_count=policy->indices_count[2];

  //if c is indexed and in-place we need to work in scratch memory
  bool inplace=(c==a || c==b);
  size_t out_size=c->stride*c->nlimbs*sizeof(xmpLimb_t);

  if(inplace) {
    xmpError_t error;
    error=xmpSetNecessaryScratchSize(handle, out_size);
    if(error!=xmpErrorSuccess)
      return error;
    xor_arguments.out_data=(xmpLimb_t*)handle->scratch;
  }

  configureActiveBlocks(handle, blocks, threads, strided_xor_kernel<GSL>);
  strided_xor_kernel<GSL><<<blocks, threads, 0, handle->stream>>>(xor_arguments, count);

  if(inplace) {
    hipMemcpyAsync(c->slimbs,xor_arguments.out_data,out_size,hipMemcpyDeviceToDevice,handle->stream);
  }

  c->setFormat(xmpFormatStrided);
  return xmpErrorSuccess;
}
//computes c=!a
xmpError_t XMPAPI xmpIntegersNot(xmpHandle_t handle, xmpIntegers_t c, const xmpIntegers_t a, uint32_t count) {
  xmpError_t error=xmpIntegersNotAsync(handle,c,a,count);
  if(error!=xmpErrorSuccess)
    return error;
  XMP_SET_DEVICE(handle);
  hipStreamSynchronize(handle->stream);
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
xmpError_t XMPAPI xmpIntegersNotAsync(xmpHandle_t handle, xmpIntegers_t c, const xmpIntegers_t a, uint32_t count) {
  dim3            blocks(DIV_ROUND_UP(count, GEOMETRY)), threads(GEOMETRY);
  int             device=handle->device;
  not_arguments_t not_arguments;
  xmpExecutionPolicy_t policy=handle->policy;

  if(c->device!=device || a->device!=device)
    return xmpErrorInvalidDevice;

  if(count==0)
    return xmpErrorSuccess;

  XMP_SET_DEVICE(handle);

  if(c->count<count)
    return xmpErrorInvalidCount;

  if(policy->indices[0] && policy->indices_count[0]<count)
    return xmpErrorInvalidCount;

  if(c->precision<a->precision)
    return xmpErrorInvalidPrecision;

  a->requireFormat(handle, xmpFormatStrided);

  // package up the arguments
  not_arguments.out_data=c->slimbs;
  not_arguments.out_len=DIV_ROUND_UP(c->precision, 32);
  not_arguments.out_stride=c->stride;
  not_arguments.a_data=a->slimbs;
  not_arguments.a_len=DIV_ROUND_UP(a->precision, 32);
  not_arguments.a_stride=a->stride;
  not_arguments.a_count=a->count;
  not_arguments.out_indices=policy->indices[0];
  not_arguments.a_indices=policy->indices[1];
  not_arguments.a_indices_count=policy->indices_count[1];

  //if c is in-place we need to work in scratch memory
  bool inplace=(c==a);
  size_t out_size=c->stride*c->nlimbs*sizeof(xmpLimb_t);

  if(inplace) {
    xmpError_t error;
    error=xmpSetNecessaryScratchSize(handle, out_size);
    if(error!=xmpErrorSuccess)
      return error;
    not_arguments.out_data=(xmpLimb_t*)handle->scratch;
  }


  configureActiveBlocks(handle, blocks, threads, strided_not_kernel<GSL>);
  strided_not_kernel<GSL><<<blocks, threads, 0, handle->stream>>>(not_arguments, count);

  if(inplace) {
    hipMemcpyAsync(c->slimbs,not_arguments.out_data,out_size,hipMemcpyDeviceToDevice,handle->stream);
  }

  c->setFormat(xmpFormatStrided);
  return xmpErrorSuccess;
}
//compute c=popc(a)
xmpError_t XMPAPI xmpIntegersPopc(xmpHandle_t handle, uint32_t *c, const xmpIntegers_t a, uint32_t count) {
  xmpError_t error=xmpIntegersPopcAsync(handle,c,a,count);
  if(error!=xmpErrorSuccess)
    return error;
  XMP_SET_DEVICE(handle);
  hipStreamSynchronize(handle->stream);
  XMP_CHECK_CUDA();
  return xmpErrorSuccess;
}
xmpError_t XMPAPI xmpIntegersPopcAsync(xmpHandle_t handle, uint32_t *c, const xmpIntegers_t a, uint32_t count) {
  dim3             blocks(DIV_ROUND_UP(count, GEOMETRY)), threads(GEOMETRY);
  int              device=handle->device;
  xmpError_t       error;
  popc_arguments_t popc_arguments;
  xmpExecutionPolicy_t policy=handle->policy;

  if(a->device!=device)
    return xmpErrorInvalidDevice;

  if(count==0)
    return xmpErrorSuccess;

  if(a->count<count)
    return xmpErrorInvalidCount;

  XMP_SET_DEVICE(handle);

  //check if we know where this pointer came from, if not assume host
  hipPointerAttribute_t attrib;
  hipError_t cerror=hipPointerGetAttributes(&attrib,c);
  if(cerror!=hipSuccess) {
    if(cerror==hipErrorInvalidValue) {
      hipGetLastError();  //reset to hipSuccess
      attrib.memoryType=hipMemoryTypeHost;
    } else {
      return xmpErrorCuda;
    }
  }

  uint32_t *dst=c;
  if(attrib.memoryType==hipMemoryTypeHost) {
    error=xmpSetNecessaryScratchSize(handle, count*sizeof(uint32_t));
    if(error!=xmpErrorSuccess)
      return error;
    dst=(uint32_t *)handle->scratch;
  }

  a->requireFormat(handle, xmpFormatStrided);

  popc_arguments.out_data=(uint32_t *)dst;
  popc_arguments.a_data=a->slimbs;
  popc_arguments.a_len=DIV_ROUND_UP(a->precision, 32);
  popc_arguments.a_stride=a->stride;
  popc_arguments.a_count=a->count;
  popc_arguments.a_indices=policy->indices[0];
  popc_arguments.a_indices_count=policy->indices_count[0];

  configureActiveBlocks(handle, blocks, threads, strided_popc_kernel<GSL>);
  strided_popc_kernel<GSL><<<blocks, threads, 0, handle->stream>>>(popc_arguments, count);

  if(attrib.memoryType==hipMemoryTypeHost) {
    hipMemcpyAsync(c,dst,sizeof(uint32_t)*count,hipMemcpyDefault,handle->stream);
  }
  return xmpErrorSuccess;
}
